#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include "signal_utils.h"

#define BLOCK_SIZE 256

// CUDA kernel for edge detection (using gradient method)
__global__ void edgeDetection(const float* inputSignal, float* outputSignal, int length) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx > 0 && idx < length - 1) {
        // Compute gradient (difference between adjacent samples)
        outputSignal[idx] = fabs(inputSignal[idx + 1] - inputSignal[idx - 1]);
    }
}

void processSignal(const std::string& inputPath, const std::string& edgeOutputPath, const std::string& restoredOutputPath) {
    std::vector<float> signal = loadSignal(inputPath);
    int length = signal.size();

    // Allocate memory for device input and output signals
    float* d_input;
    float* d_output;

    hipMalloc(&d_input, length * sizeof(float));
    hipMalloc(&d_output, length * sizeof(float));

    // Copy the signal to the device
    hipMemcpy(d_input, signal.data(), length * sizeof(float), hipMemcpyHostToDevice);

    // Set up grid and block size for CUDA kernel
    int gridSize = (length + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Run the edge detection kernel
    edgeDetection<<<gridSize, BLOCK_SIZE>>>(d_input, d_output, length);
    hipDeviceSynchronize();

    // Copy the result back to host
    std::vector<float> edgeSignal(length);
    hipMemcpy(edgeSignal.data(), d_output, length * sizeof(float), hipMemcpyDeviceToHost);

    // Save the edge-detected signal
    saveSignal(edgeSignal, edgeOutputPath);

    // Restore the signal by simple moving average (smoothing)
    std::vector<float> restoredSignal = restoreSignal(edgeSignal);

    // Save the restored signal
    saveSignal(restoredSignal, restoredOutputPath);

    // Clean up
    hipFree(d_input);
    hipFree(d_output);
}

int main() {
    processSignal("data/noisy_signal.csv", "output/edge_detected_signal.csv", "output/restored_signal.csv");
    std::cout << "Processing completed.\n";
    return 0;
}
